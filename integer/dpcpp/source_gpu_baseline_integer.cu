    
#include <hip/hip_runtime.h>
#include <iostream>
    #include <math.h>
    #include <chrono>     
    #include <algorithm>

	using namespace std;
	using namespace std::chrono;





    // CUDA kernel to add elements of two arrays
    __global__
    void add(int n, int *x, int *y)
    {
      int index = blockIdx.x * blockDim.x + threadIdx.x;
      int stride = blockDim.x * gridDim.x;
      for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
    }
     
    int main(void)
    {

  

	std::chrono::time_point<std::chrono::system_clock> start,stop;
	start = std::chrono::system_clock::now();;
	
	int count = 64;

	while (count > 0){ 

	int N = 1<<27;
      	int *x, *y;
     
      // Allocate Unified Memory -- accessible from CPU or GPU
      hipMallocManaged(&x, N*sizeof(int));
      hipMallocManaged(&y, N*sizeof(int));
     
      // initialize x and y arrays on the host
      for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
      }

      // Launch kernel on 1M elements on the GPU
      int blockSize = 256;
      int numBlocks = (N + blockSize - 1) / blockSize;
      add<<<numBlocks, blockSize>>>(N, x, y);

      // Wait for GPU to finish before accessing on host
      hipDeviceSynchronize();

/*      // Check for errors (all values should be 3.0f)
      float maxError = 0.0f;
      for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));

	if(count<2){	std::cout << "Max error: " << maxError << std::endl;}
*/
      // Free memory
      hipFree(x);
      hipFree(y);

	count--;
	}
	stop = std::chrono::system_clock::now();


	std::chrono::duration<double> elapsed_seconds = stop - start;


	cout << "	Time taken by program:		 " << elapsed_seconds.count() << "s\n";

	 return 0;
    }
