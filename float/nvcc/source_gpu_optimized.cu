    #include <iostream>
    #include <math.h>
    #include <chrono>
    #include <algorithm>
    #include "hip/hip_runtime.h"
    #include "hip/hip_fp16.h"
    

        using namespace std;
        using namespace std::chrono;


    // CUDA kernel to add elements of two arrays
 __global__
    void add(int n, float*x, float*y)
    {
      int index = blockIdx.x * blockDim.x + threadIdx.x;
      int stride = blockDim.x * gridDim.x;
//	 for (int i = index; i < n; i += stride)
//	y[i] = __device__ ​ __half __hadd ( const __half x[i], const __half y[i] ) ;  

	 for (int i = index; i < n; i += stride)
		 y[i] = x[i] + y[i];
    }


__global__
	void init(int n, float*x, float*y) 
{
  	int index = threadIdx.x + blockIdx.x * blockDim.x;
  	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) 
	{
    		x[i] = 1.0f;
    		y[i] = 2.0f;
  	}
}     




    int main(void)
    {

	//Start counting

	std::chrono::time_point<std::chrono::system_clock> start,stop;
        start = std::chrono::system_clock::now();;

	int count =64;

	while (count >0){
      int N = 1<<27;
	 float *x, *y;
     
      // Allocate Unified Memory -- accessible from CPU or GPU
      hipMallocManaged(&x, N*sizeof(float));
      hipMallocManaged(&y, N*sizeof(float));
     
      
//define Blocks and strides
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;


// initialize x and y arrays on the host


	init<<<numBlocks, blockSize>>>(N,x,y);

     
      // Launch kernel on 1M elements on the GPU
      
	add<<<numBlocks, blockSize>>>(N, x, y);
     
      // Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

      // Check for errors (all values should be 3.0f)
      float maxError = 0.0f;
      for (int i = 0; i < N; i++)
	maxError = fmax(maxError, fabs(y[i]-3.0f));
	if(count<2){    std::cout << "Max error: " << maxError << std::endl;}
     
      // Free memory
      hipFree(x);
      hipFree(y);
	count--;
	}
     // calculate and output time 
	stop = std::chrono::system_clock::now();
        std::chrono::duration<float> elapsed_seconds = stop - start;

//	std::cout << "Max error: " << maxError << std::endl;

        cout << "	Time taken by program:		 " << elapsed_seconds.count() << "s\n";
      return 0;
    }
